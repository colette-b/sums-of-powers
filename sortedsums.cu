#include "hip/hip_runtime.h"
#include <random>
#include <limits>
#include <type_traits>
#include <functional>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/copy.h>
#include <thrust/random.h>
#include <thrust/binary_search.h>

#include "basic.cu"
#include "logging.cc"

/*  input: 
 *  A, B are two arrays (of int-like type),
 *  A_param, B_param are two arrays (of any types T, S),
 *  interval [L, H),
 *  some bool function condition(T t, S s),
 *  output:
 *  all collisions in the interval, where we iterate over all
 *  sums A[i] + B[j] such that condition(A_param[i], B_param[j]) holds.
 */

class range_too_large_error : public std::runtime_error {
    public:
    range_too_large_error() : std::runtime_error("total_deposit_size > MAX_BATCH_SIZE") { }
};

template<typename data_t, typename Aparam_t, typename Bparam_t>
struct SortedSumsPointers {
    size_t A_size, B_size;
    data_t *A, *B;
    void *items;
    Aparam_t *A_param;
    Bparam_t *B_param;
    int *lowerbounds, *prefix_sums;
};

template<typename data_t, typename Aparam_t, typename Bparam_t, typename Condition>
__global__
void precount(data_t H, SortedSumsPointers<data_t, Aparam_t, Bparam_t> ssp) {
    int count = 0;
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= ssp.A_size)
        return;
    data_t diff = H - ssp.A[i];
    for(int j = ssp.lowerbounds[i]; j < ssp.B_size && ssp.B[j] < diff && ssp.B[j] >= ssp.A[i]; j++) {
        if(Condition::condition(ssp.A_param[i], ssp.B_param[j])) {
            count++;
        }
    }
    ssp.prefix_sums[i] = count;
}

__device__
unsigned long long quickhash(__int128_t x) {
    unsigned long long arr[6] = {
        8497707237685197884ULL,
        4712945162569964047ULL,
        1811193290306197184ULL,
        1346773749745882465ULL,
        1657195945765243368ULL,
        1454561834835265983ULL
    };
    __int128_t *coefs = reinterpret_cast<__int128_t*>(arr);
    __int128_t xlow = x&((__int128_t(1)<<64) - 1), xhigh = x>>64;
    return (coefs[0] * xlow + coefs[1] * xhigh + coefs[2]) >> 64;
}

template<typename data_t>
struct DepositHashed {
    using deposit_t = unsigned long long;
    __device__
    static void deposit_value(deposit_t& item, data_t x, data_t y) {
        item = quickhash(x + y);
    }
};

template<typename data_t>
struct DepositUnhashed {
    using deposit_t = data_t;
    __device__
    static void deposit_value(deposit_t& item, data_t x, data_t y) {
        item = x + y;
    }
};

template<typename data_t>
struct DepositBothSummands {
    using deposit_t = std::pair<data_t, data_t>;
    __device__
    static void deposit_value(deposit_t& item, data_t x, data_t y) {
        item.first = x;
        item.second = y;
    }
};

template<typename Deposit, typename data_t, typename Aparam_t, typename Bparam_t, typename Condition>
__global__
void deposit(SortedSumsPointers<data_t, Aparam_t, Bparam_t> ssp) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i >= ssp.A_size)
        return;
    int count = ssp.prefix_sums[i];
    int finish = ssp.prefix_sums[i + 1];
    typename Deposit::deposit_t *ptr = reinterpret_cast<Deposit::deposit_t*>(ssp.items);
    for(int j = ssp.lowerbounds[i]; count < finish; j++) {
        if(Condition::condition(ssp.A_param[i], ssp.B_param[j])) {
            Deposit::deposit_value(ptr[count], ssp.A[i], ssp.B[j]);
            count++;
        }
    }
}

template<typename data_t, typename bool_t>
__global__
void check_consecutive_eq(bool_t *raw_eq_check, data_t *raw_items) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    raw_eq_check[i] = (raw_items[i] == raw_items[i+1]);
}

template<typename data_t, typename Aparam_t, typename Bparam_t, typename Condition>
std::vector<std::pair<data_t, data_t>> restore(thrust::host_vector<data_t> A, thrust::host_vector<data_t> B, 
                                        thrust::host_vector<Aparam_t> Aparam, thrust::host_vector<Bparam_t> Bparam, data_t collision) {
    int l = 0, r = B.size() - 1;
    std::vector<std::pair<data_t, data_t>> values;
    while(l < A.size() and r >= 0) {
        if(A[l] + B[r] < collision) {
            l++;
            continue;
        }
        if(A[l] + B[r] > collision) {
            r--;
            continue;
        }
        if(A[l] + B[r] == collision) {
            if(Condition::condition(Aparam[l], Bparam[r])) {
                values.push_back(std::make_pair(A[l], B[r]));
            }
            l++;
            r--;
        }
    }
    return values;
}

template<typename data_t, typename Aparam_t, typename Bparam_t, typename Condition>
class SortedSums {
    int A_size, B_size;
    int MAX_BATCH_SIZE, MAX_EXPECTED_COLLISIONS;
    thrust::device_vector<data_t> &A, &B;
    thrust::device_vector<data_t> items, lowerbound_args, collisions_collected;
    thrust::device_vector<Aparam_t> &A_param;
    thrust::device_vector<Bparam_t> &B_param;
    thrust::device_vector<int> prefix_sums, lowerbounds, eq_check;

    public:
    SortedSums(
        int MAX_BATCH_SIZE,
        int MAX_EXPECTED_COLLISIONS,
        thrust::device_vector<data_t> &A, 
        thrust::device_vector<data_t> &B,
        thrust::device_vector<Aparam_t> &A_param,
        thrust::device_vector<Bparam_t> &B_param
    ) : 
        A(A), B(B), A_param(A_param), B_param(B_param),
        MAX_BATCH_SIZE(MAX_BATCH_SIZE),
        MAX_EXPECTED_COLLISIONS(MAX_EXPECTED_COLLISIONS),
        A_size(A.size()), B_size(B.size()),
        prefix_sums(A_size + 1),
        lowerbound_args(A_size),
        lowerbounds(A_size),
        items(MAX_BATCH_SIZE + 1),
        eq_check(MAX_BATCH_SIZE),
        collisions_collected(MAX_EXPECTED_COLLISIONS)
    { }

    SortedSumsPointers<data_t, Aparam_t, Bparam_t> get_ssp() {
        SortedSumsPointers<data_t, Aparam_t, Bparam_t> ssp;
        ssp.A = thrust::raw_pointer_cast(A.data());
        ssp.B = thrust::raw_pointer_cast(B.data());
        ssp.A_param = thrust::raw_pointer_cast(A_param.data());
        ssp.B_param = thrust::raw_pointer_cast(B_param.data());
        ssp.items = reinterpret_cast<void*>(thrust::raw_pointer_cast(items.data()));
        ssp.A_size = A.size();
        ssp.B_size = B.size();
        ssp.lowerbounds = thrust::raw_pointer_cast(lowerbounds.data());
        ssp.prefix_sums = thrust::raw_pointer_cast(prefix_sums.data());
        return ssp;
    }

    int make_prefix_sums(data_t L, data_t H, SpecializedLogger& fcl) {
        // prepares the prefix_sums vector
        // returns the total deposit size
        fcl.time_tick();
        thrust::fill(lowerbound_args.begin(), lowerbound_args.end(), L);
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        fcl.time_tick();
        thrust::transform(lowerbound_args.begin(), lowerbound_args.end(), A.begin(), lowerbound_args.begin(), thrust::minus<data_t>());
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        fcl.time_tick();
        thrust::lower_bound(B.begin(), B.end(), lowerbound_args.begin(), lowerbound_args.end(), lowerbounds.begin());
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        fcl.time_tick();
        precount<data_t, Aparam_t, Bparam_t, Condition>
                <<<1 + A_size/GPU_BLOCK_SIZE, GPU_BLOCK_SIZE>>>
                (H, get_ssp());
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        int total_deposit_size = thrust::reduce(prefix_sums.begin(), prefix_sums.end() - 1);
        thrust::exclusive_scan(prefix_sums.begin(), prefix_sums.end(), prefix_sums.begin());
        return total_deposit_size;
    }
    
    template<typename Deposit, bool tick>
    int check_collisions(data_t L, data_t H, int total_deposit_size, SpecializedLogger& fcl) {
        if(tick)
            fcl.time_tick();
        deposit<Deposit, data_t, Aparam_t, Bparam_t, Condition>
               <<<1 + A_size/GPU_BLOCK_SIZE, GPU_BLOCK_SIZE>>>(get_ssp());
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        if(tick)
            fcl.time_tick();
        typename Deposit::deposit_t *items_ptr = 
            reinterpret_cast<Deposit::deposit_t*>(thrust::raw_pointer_cast(items.data()));
        thrust::sort(thrust::device, items_ptr, items_ptr + total_deposit_size);
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        if(tick)
            fcl.time_tick();
        check_consecutive_eq<<<(total_deposit_size + GPU_BLOCK_SIZE - 1)/GPU_BLOCK_SIZE, GPU_BLOCK_SIZE>>>(
            thrust::raw_pointer_cast(eq_check.data()), 
            items_ptr
        );
        int collision_count = thrust::reduce(eq_check.begin(), eq_check.begin() + total_deposit_size - 1);
        return collision_count;
    }

    std::vector<std::pair<data_t, data_t>> restore_collision(data_t P) {
        SpecializedLogger dummy;
        int total_deposit_size = make_prefix_sums(P, P + 1, dummy);
        deposit<DepositBothSummands<data_t>, data_t, Aparam_t, Bparam_t, Condition>
               <<<1 + A_size/GPU_BLOCK_SIZE, GPU_BLOCK_SIZE>>>(get_ssp());
        std::vector<std::pair<data_t, data_t>> vec(total_deposit_size);
        hipMemcpy(
            vec.data(), 
            get_ssp().items, 
            sizeof(std::pair<data_t, data_t>) * total_deposit_size,
            hipMemcpyDeviceToHost
        );
        return vec;
    }

    int check_range(data_t L, data_t H, SpecializedLogger& fcl, std::function<void(data_t)> do_on_collision) {
        int total_deposit_size = make_prefix_sums(L, H, fcl);
        if(total_deposit_size > MAX_BATCH_SIZE) {
            fcl.cleanup();
            throw range_too_large_error();
        }
        if(total_deposit_size <= 1) {
            fcl.cleanup();
            return total_deposit_size;
        }
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());

        int collision_count = check_collisions<DepositHashed<data_t>, true>(L, H, total_deposit_size, fcl);
        if(collision_count) {
            std::cerr << "seen " << collision_count << " quick collisions\n";
            collision_count = check_collisions<DepositUnhashed<data_t>, false>(L, H, total_deposit_size, fcl);
        }

        fcl.time_tick();
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        thrust::host_vector<data_t> h_collisions_collected(0);
        if(collision_count > 0) {
            thrust::copy_if(
                items.begin(), 
                items.begin() + total_deposit_size - 1, 
                eq_check.begin(), 
                collisions_collected.begin(), 
                thrust::identity<bool>()
            );
            h_collisions_collected = collisions_collected;
            for(int i = 0; i < collision_count; i++) {
                do_on_collision(h_collisions_collected[i]);
            }
        }
        fcl.time_tick();
        fcl.set<0>(total_deposit_size);
        gpuErrchk(hipPeekAtLastError()); gpuErrchk(hipDeviceSynchronize());
        return total_deposit_size;
    }

    template<typename Logger>
    size_t check_large_range(data_t L, data_t H, Logger& fcl, std::function<void(data_t)> do_on_collision) {
        data_t current_L = L, jump = 1 << 20;
        size_t total = 0;
        for(int iter = 0; current_L < H; iter++){
            try {
                data_t current_H = std::min(current_L + jump, H);
                int batch_size = check_range(current_L, current_H, fcl, do_on_collision);
                if(iter % 10 == 9) {
                    std::cerr << "[" << current_L << ", " << current_H << ")\t";
                    fcl.show();
                }
                if(batch_size < 0.1 * MAX_BATCH_SIZE) {
                    jump *= 2;
                }
                if(batch_size < 0.5 * MAX_BATCH_SIZE) {
                    jump *= 1.05;
                }
                total += batch_size;
                current_L = current_H;
            } catch (range_too_large_error& e) {
                std::cout << "too many items in range; decreasing jump\n";
                jump /= 1.3;
            }
        }
        return total;
    }
};
